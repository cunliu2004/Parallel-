#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define BLOCK_DIM 16



__global__ void compute_distances(float * ref,
                                  int     ref_width,
                                  int     ref_pitch,
                                  float * query,
                                  int     query_width,
                                  int     query_pitch,
                                  int     height,
                                  float * dist) {

  
    __shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
    __shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

  
    __shared__ int begin_A;
    __shared__ int begin_B;
    __shared__ int step_A;
    __shared__ int step_B;
    __shared__ int end_A;


    int tx = threadIdx.x;
    int ty = threadIdx.y;

   
    float ssd = 0.f;


    begin_A = BLOCK_DIM * blockIdx.y;
    begin_B = BLOCK_DIM * blockIdx.x;
    step_A  = BLOCK_DIM * ref_pitch;
    step_B  = BLOCK_DIM * query_pitch;
    end_A   = begin_A + (height-1) * ref_pitch;

  
    int cond0 = (begin_A + tx < ref_width); 
    int cond1 = (begin_B + tx < query_width); 
    int cond2 = (begin_A + ty < ref_width); 

 
    for (int a = begin_A, b = begin_B; a <= end_A; a += step_A, b += step_B) {

   
        if (a/ref_pitch + ty < height) {
            shared_A[ty][tx] = (cond0)? ref[a + ref_pitch * ty + tx] : 0;
            shared_B[ty][tx] = (cond1)? query[b + query_pitch * ty + tx] : 0;
        }
        else {
            shared_A[ty][tx] = 0;
            shared_B[ty][tx] = 0;
        }

   
        __syncthreads();

        if (cond2 && cond1) {
            for (int k = 0; k < BLOCK_DIM; ++k){
                float tmp = shared_A[k][ty] - shared_B[k][tx];
                ssd += tmp*tmp;
            }
        }

       
        __syncthreads();
    }

 
    if (cond2 && cond1) {
        dist[ (begin_A + ty) * query_pitch + begin_B + tx ] = ssd;
    }
}



__global__ void compute_distance_texture(hipTextureObject_t ref,
                                         int                 ref_width,
                                         float *             query,
                                         int                 query_width,
                                         int                 query_pitch,
                                         int                 height,
                                         float*              dist) {
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if ( xIndex<query_width && yIndex<ref_width) {
        float ssd = 0.f;
        for (int i=0; i<height; i++) {
            float tmp  = tex2D<float>(ref, (float)yIndex, (float)i) - query[i * query_pitch + xIndex];
            ssd += tmp * tmp;
        }
        dist[yIndex * query_pitch + xIndex] = ssd;
    }
}



__global__ void modified_insertion_sort(float * dist,
                                        int     dist_pitch,
                                        int *   index,
                                        int     index_pitch,
                                        int     width,
                                        int     height,
                                        int     k){


    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;


    if (xIndex < width) {

        float * p_dist  = dist  + xIndex;
        int *   p_index = index + xIndex;


        p_index[0] = 0;

 
        for (int i=1; i<height; ++i) {


            float curr_dist = p_dist[i*dist_pitch];
            int   curr_index  = i;

 
            if (i >= k && curr_dist >= p_dist[(k-1)*dist_pitch]) {
                continue;
            }


            int j = min(i, k-1);
            while (j > 0 && p_dist[(j-1)*dist_pitch] > curr_dist) {
                p_dist[j*dist_pitch]   = p_dist[(j-1)*dist_pitch];
                p_index[j*index_pitch] = p_index[(j-1)*index_pitch];
                --j;
            }

 
            p_dist[j*dist_pitch]   = curr_dist;
            p_index[j*index_pitch] = curr_index; 
        }
    }
}



__global__ void compute_sqrt(float * dist, int width, int pitch, int k){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if (xIndex<width && yIndex<k)
        dist[yIndex*pitch + xIndex] = sqrt(dist[yIndex*pitch + xIndex]);
}



__global__ void compute_squared_norm(float * array, int width, int pitch, int height, float * norm){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (xIndex<width){
        float sum = 0.f;
        for (int i=0; i<height; i++){
            float val = array[i*pitch+xIndex];
            sum += val*val;
        }
        norm[xIndex] = sum;
    }
}



__global__ void add_reference_points_norm(float * array, int width, int pitch, int height, float * norm){
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    unsigned int xIndex = blockIdx.x * blockDim.x + tx;
    unsigned int yIndex = blockIdx.y * blockDim.y + ty;
    __shared__ float shared_vec[16];
    if (tx==0 && yIndex<height)
        shared_vec[ty] = norm[yIndex];
    __syncthreads();
    if (xIndex<width && yIndex<height)
        array[yIndex*pitch+xIndex] += shared_vec[ty];
}



__global__ void add_query_points_norm_and_sqrt(float * array, int width, int pitch, int k, float * norm){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if (xIndex<width && yIndex<k)
        array[yIndex*pitch + xIndex] = sqrt(array[yIndex*pitch + xIndex] + norm[xIndex]);
}


bool knn_cuda_global(const float * ref,
                     int           ref_nb,
                     const float * query,
                     int           query_nb,
                     int           dim,
                     int           k,
                     float *       knn_dist,
                     int *         knn_index) {


    const unsigned int size_of_float = sizeof(float);
    const unsigned int size_of_int   = sizeof(int);


    hipError_t err0, err1, err2, err3;

    int nb_devices;
    err0 = hipGetDeviceCount(&nb_devices);
    if (err0 != hipSuccess || nb_devices == 0) {
        printf("ERROR: No CUDA device found\n");
        return false;
    }


    err0 = hipSetDevice(0);
    if (err0 != hipSuccess) {
        printf("ERROR: Cannot set the chosen CUDA device\n");
        return false;
    }


    float * ref_dev   = NULL;
    float * query_dev = NULL;
    float * dist_dev  = NULL;
    int   * index_dev = NULL;
    size_t  ref_pitch_in_bytes;
    size_t  query_pitch_in_bytes;
    size_t  dist_pitch_in_bytes;
    size_t  index_pitch_in_bytes;
    err0 = hipMallocPitch((void**)&ref_dev,   &ref_pitch_in_bytes,   ref_nb   * size_of_float, dim);
    err1 = hipMallocPitch((void**)&query_dev, &query_pitch_in_bytes, query_nb * size_of_float, dim);
    err2 = hipMallocPitch((void**)&dist_dev,  &dist_pitch_in_bytes,  query_nb * size_of_float, ref_nb);
    err3 = hipMallocPitch((void**)&index_dev, &index_pitch_in_bytes, query_nb * size_of_int,   k);
    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess) {
        printf("ERROR: Memory allocation error\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }


    size_t ref_pitch   = ref_pitch_in_bytes   / size_of_float;
    size_t query_pitch = query_pitch_in_bytes / size_of_float;
    size_t dist_pitch  = dist_pitch_in_bytes  / size_of_float;
    size_t index_pitch = index_pitch_in_bytes / size_of_int;


    if (query_pitch != dist_pitch || query_pitch != index_pitch) {
        printf("ERROR: Invalid pitch value\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false; 
    }


    err0 = hipMemcpy2D(ref_dev,   ref_pitch_in_bytes,   ref,   ref_nb * size_of_float,   ref_nb * size_of_float,   dim, hipMemcpyHostToDevice);
    err1 = hipMemcpy2D(query_dev, query_pitch_in_bytes, query, query_nb * size_of_float, query_nb * size_of_float, dim, hipMemcpyHostToDevice);
    if (err0 != hipSuccess || err1 != hipSuccess) {
        printf("ERROR: Unable to copy data from host to device\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false; 
    }


    dim3 block0(BLOCK_DIM, BLOCK_DIM, 1);
    dim3 grid0(query_nb / BLOCK_DIM, ref_nb / BLOCK_DIM, 1);
    if (query_nb % BLOCK_DIM != 0) grid0.x += 1;
    if (ref_nb   % BLOCK_DIM != 0) grid0.y += 1;
    compute_distances<<<grid0, block0>>>(ref_dev, ref_nb, ref_pitch, query_dev, query_nb, query_pitch, dim, dist_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }


    dim3 block1(256, 1, 1);
    dim3 grid1(query_nb / 256, 1, 1);
    if (query_nb % 256 != 0) grid1.x += 1;
    modified_insertion_sort<<<grid1, block1>>>(dist_dev, dist_pitch, index_dev, index_pitch, query_nb, ref_nb, k);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }


    dim3 block2(16, 16, 1);
    dim3 grid2(query_nb / 16, k / 16, 1);
    if (query_nb % 16 != 0) grid2.x += 1;
    if (k % 16 != 0)        grid2.y += 1;
    compute_sqrt<<<grid2, block2>>>(dist_dev, query_nb, query_pitch, k);	
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }


    err0 = hipMemcpy2D(knn_dist,  query_nb * size_of_float, dist_dev,  dist_pitch_in_bytes,  query_nb * size_of_float, k, hipMemcpyDeviceToHost);
    err1 = hipMemcpy2D(knn_index, query_nb * size_of_int,   index_dev, index_pitch_in_bytes, query_nb * size_of_int,   k, hipMemcpyDeviceToHost);
    if (err0 != hipSuccess || err1 != hipSuccess) {
        printf("ERROR: Unable to copy data from device to host\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false; 
    }

 
    hipFree(ref_dev);
    hipFree(query_dev);
    hipFree(dist_dev);
    hipFree(index_dev); 

    return true;
}


bool knn_cuda_texture(const float * ref,
                      int           ref_nb,
                      const float * query,
                      int           query_nb,
                      int           dim,
                      int           k,
                      float *       knn_dist,
                      int *         knn_index) {


    unsigned int size_of_float = sizeof(float);
    unsigned int size_of_int   = sizeof(int);   

 
    hipError_t err0, err1, err2;

 
    int nb_devices;
    err0 = hipGetDeviceCount(&nb_devices);
    if (err0 != hipSuccess || nb_devices == 0) {
        printf("ERROR: No CUDA device found\n");
        return false;
    }

  
    err0 = hipSetDevice(0);
    if (err0 != hipSuccess) {
        printf("ERROR: Cannot set the chosen CUDA device\n");
        return false;
    }


    float * query_dev = NULL;
    float * dist_dev  = NULL;
    int *   index_dev = NULL;
    size_t  query_pitch_in_bytes;
    size_t  dist_pitch_in_bytes;
    size_t  index_pitch_in_bytes;
    err0 = hipMallocPitch((void**)&query_dev, &query_pitch_in_bytes, query_nb * size_of_float, dim);
    err1 = hipMallocPitch((void**)&dist_dev,  &dist_pitch_in_bytes,  query_nb * size_of_float, ref_nb);
    err2 = hipMallocPitch((void**)&index_dev, &index_pitch_in_bytes, query_nb * size_of_int,   k);
    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess) {
        printf("ERROR: Memory allocation error (cudaMallocPitch)\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }

 
    size_t query_pitch = query_pitch_in_bytes / size_of_float;
    size_t dist_pitch  = dist_pitch_in_bytes  / size_of_float;
    size_t index_pitch = index_pitch_in_bytes / size_of_int;


    if (query_pitch != dist_pitch || query_pitch != index_pitch) {
        printf("ERROR: Invalid pitch value\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false; 
    }


    err0 = hipMemcpy2D(query_dev, query_pitch_in_bytes, query, query_nb * size_of_float, query_nb * size_of_float, dim, hipMemcpyHostToDevice);
    if (err0 != hipSuccess) {
        printf("ERROR: Unable to copy data from host to device\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);        
        return false; 
    }


    hipArray* ref_array_dev = NULL;
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    err0 = hipMallocArray(&ref_array_dev, &channel_desc, ref_nb, dim);
    if (err0 != hipSuccess) {
        printf("ERROR: Memory allocation error (cudaMallocArray)\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        return false; 
    }


    err0 = hipMemcpyToArray(ref_array_dev, 0, 0, ref, ref_nb * size_of_float * dim, hipMemcpyHostToDevice);
    if (err0 != hipSuccess) {
        printf("ERROR: Unable to copy data from host to device\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFreeArray(ref_array_dev);
        return false; 
    }


    struct hipResourceDesc res_desc;
    memset(&res_desc, 0, sizeof(res_desc));
    res_desc.resType         = hipResourceTypeArray;
    res_desc.res.array.array = ref_array_dev;


    struct hipTextureDesc tex_desc;
    memset(&tex_desc, 0, sizeof(tex_desc));
    tex_desc.addressMode[0]   = hipAddressModeClamp;
    tex_desc.addressMode[1]   = hipAddressModeClamp;
    tex_desc.filterMode       = hipFilterModePoint;
    tex_desc.readMode         = hipReadModeElementType;
    tex_desc.normalizedCoords = 0;


    hipTextureObject_t ref_tex_dev = 0;
    err0 = hipCreateTextureObject(&ref_tex_dev, &res_desc, &tex_desc, NULL);
    if (err0 != hipSuccess) {
        printf("ERROR: Unable to create the texture\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFreeArray(ref_array_dev);
        return false; 
    }


    dim3 block0(16, 16, 1);
    dim3 grid0(query_nb / 16, ref_nb / 16, 1);
    if (query_nb % 16 != 0) grid0.x += 1;
    if (ref_nb   % 16 != 0) grid0.y += 1;
    compute_distance_texture<<<grid0, block0>>>(ref_tex_dev, ref_nb, query_dev, query_nb, query_pitch, dim, dist_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFreeArray(ref_array_dev);
        hipDestroyTextureObject(ref_tex_dev);
        return false;
    }


    dim3 block1(256, 1, 1);
    dim3 grid1(query_nb / 256, 1, 1);
    if (query_nb % 256 != 0) grid1.x += 1;
    modified_insertion_sort<<<grid1, block1>>>(dist_dev, dist_pitch, index_dev, index_pitch, query_nb, ref_nb, k);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFreeArray(ref_array_dev);
        hipDestroyTextureObject(ref_tex_dev);
        return false;
    }


    dim3 block2(16, 16, 1);
    dim3 grid2(query_nb / 16, k / 16, 1);
    if (query_nb % 16 != 0) grid2.x += 1;
    if (k % 16 != 0)        grid2.y += 1;
    compute_sqrt<<<grid2, block2>>>(dist_dev, query_nb, query_pitch, k);	
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFreeArray(ref_array_dev);
        hipDestroyTextureObject(ref_tex_dev);
        return false;
    }


    err0 = hipMemcpy2D(knn_dist,  query_nb * size_of_float, dist_dev,  dist_pitch_in_bytes,  query_nb * size_of_float, k, hipMemcpyDeviceToHost);
    err1 = hipMemcpy2D(knn_index, query_nb * size_of_int,   index_dev, index_pitch_in_bytes, query_nb * size_of_int,   k, hipMemcpyDeviceToHost);
    if (err0 != hipSuccess || err1 != hipSuccess) {
        printf("ERROR: Unable to copy data from device to host\n");
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFreeArray(ref_array_dev);
        hipDestroyTextureObject(ref_tex_dev);
        return false; 
    }


    hipFree(query_dev);
    hipFree(dist_dev);
    hipFree(index_dev);
    hipFreeArray(ref_array_dev);
    hipDestroyTextureObject(ref_tex_dev);

    return true;
}


bool knn_cublas(const float * ref,
                int           ref_nb,
                const float * query,
                int           query_nb,
                int           dim, 
                int           k, 
                float *       knn_dist,
                int *         knn_index) {


    const unsigned int size_of_float = sizeof(float);
    const unsigned int size_of_int   = sizeof(int);


    hipError_t  err0, err1, err2, err3, err4, err5;


    int nb_devices;
    err0 = hipGetDeviceCount(&nb_devices);
    if (err0 != hipSuccess || nb_devices == 0) {
        printf("ERROR: No CUDA device found\n");
        return false;
    }


    err0 = hipSetDevice(0);
    if (err0 != hipSuccess) {
        printf("ERROR: Cannot set the chosen CUDA device\n");
        return false;
    }


    cublasInit();

    // Allocate global memory
    float * ref_dev        = NULL;
    float * query_dev      = NULL;
    float * dist_dev       = NULL;
    int   * index_dev      = NULL;
    float * ref_norm_dev   = NULL;
    float * query_norm_dev = NULL;
    size_t  ref_pitch_in_bytes;
    size_t  query_pitch_in_bytes;
    size_t  dist_pitch_in_bytes;
    size_t  index_pitch_in_bytes;
    err0 = hipMallocPitch((void**)&ref_dev,   &ref_pitch_in_bytes,   ref_nb   * size_of_float, dim);
    err1 = hipMallocPitch((void**)&query_dev, &query_pitch_in_bytes, query_nb * size_of_float, dim);
    err2 = hipMallocPitch((void**)&dist_dev,  &dist_pitch_in_bytes,  query_nb * size_of_float, ref_nb);
    err3 = hipMallocPitch((void**)&index_dev, &index_pitch_in_bytes, query_nb * size_of_int,   k);
    err4 = hipMalloc((void**)&ref_norm_dev,   ref_nb   * size_of_float);
    err5 = hipMalloc((void**)&query_norm_dev, query_nb * size_of_float);
    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess || err5 != hipSuccess) {
        printf("ERROR: Memory allocation error\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false;
    }


    size_t ref_pitch   = ref_pitch_in_bytes   / size_of_float;
    size_t query_pitch = query_pitch_in_bytes / size_of_float;
    size_t dist_pitch  = dist_pitch_in_bytes  / size_of_float;
    size_t index_pitch = index_pitch_in_bytes / size_of_int;


    if (query_pitch != dist_pitch || query_pitch != index_pitch) {
        printf("ERROR: Invalid pitch value\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false; 
    }


    err0 = hipMemcpy2D(ref_dev,   ref_pitch_in_bytes,   ref,   ref_nb * size_of_float,   ref_nb * size_of_float,   dim, hipMemcpyHostToDevice);
    err1 = hipMemcpy2D(query_dev, query_pitch_in_bytes, query, query_nb * size_of_float, query_nb * size_of_float, dim, hipMemcpyHostToDevice);
    if (err0 != hipSuccess || err1 != hipSuccess) {
        printf("ERROR: Unable to copy data from host to device\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false; 
    }


    dim3 block0(256, 1, 1);
    dim3 grid0(ref_nb / 256, 1, 1);
    if (ref_nb % 256 != 0) grid0.x += 1;
    compute_squared_norm<<<grid0, block0>>>(ref_dev, ref_nb, ref_pitch, dim, ref_norm_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false;
    }


    dim3 block1(256, 1, 1);
    dim3 grid1(query_nb / 256, 1, 1);
    if (query_nb % 256 != 0) grid1.x += 1;
    compute_squared_norm<<<grid1, block1>>>(query_dev, query_nb, query_pitch, dim, query_norm_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false;
    }


    cublasSgemm('n', 't', (int)query_pitch, (int)ref_pitch, dim, (float)-2.0, query_dev, query_pitch, ref_dev, ref_pitch, (float)0.0, dist_dev, query_pitch);
    if (cublasGetError() != HIPBLAS_STATUS_SUCCESS) {
        printf("ERROR: Unable to execute cublasSgemm\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false;       
    }


    dim3 block2(16, 16, 1);
    dim3 grid2(query_nb / 16, ref_nb / 16, 1);
    if (query_nb % 16 != 0) grid2.x += 1;
    if (ref_nb   % 16 != 0) grid2.y += 1;
    add_reference_points_norm<<<grid2, block2>>>(dist_dev, query_nb, dist_pitch, ref_nb, ref_norm_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false;
    }


    modified_insertion_sort<<<grid1, block1>>>(dist_dev, dist_pitch, index_dev, index_pitch, query_nb, ref_nb, k);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false;
    }


    dim3 block3(16, 16, 1);
    dim3 grid3(query_nb / 16, k / 16, 1);
    if (query_nb % 16 != 0) grid3.x += 1;
    if (k        % 16 != 0) grid3.y += 1;
    add_query_points_norm_and_sqrt<<<grid3, block3>>>(dist_dev, query_nb, dist_pitch, k, query_norm_dev);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false;
    }


    err0 = hipMemcpy2D(knn_dist,  query_nb * size_of_float, dist_dev,  dist_pitch_in_bytes,  query_nb * size_of_float, k, hipMemcpyDeviceToHost);
    err1 = hipMemcpy2D(knn_index, query_nb * size_of_int,   index_dev, index_pitch_in_bytes, query_nb * size_of_int,   k, hipMemcpyDeviceToHost);
    if (err0 != hipSuccess || err1 != hipSuccess) {
        printf("ERROR: Unable to copy data from device to host\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev);
        hipFree(ref_norm_dev);
        hipFree(query_norm_dev);
        cublasShutdown();
        return false; 
    }


    hipFree(ref_dev);
    hipFree(query_dev);
    hipFree(dist_dev);
    hipFree(index_dev);
    hipFree(ref_norm_dev);
    hipFree(query_norm_dev);
    cublasShutdown();

    return true;
}
